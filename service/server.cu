#include <unistd.h>
#include "helper.cuh"
#include <algorithm>
#include <boost/property_tree/ptree.hpp>
#include <boost/property_tree/json_parser.hpp>
#include <RemoteCommunication.hh>
#include <threadpool.hh>
#include <csignal>

// CJD218: Unused currently
#include <faulttolerance/fault_tolerance.h>

namespace pt = boost::property_tree;
using BatchWrapper = std::vector<RequestWrapper<unsigned long long, data_t *>>;
//#ifdef MODEL_CHANGE
using Model = kvgpu::AnalyticalModel<unsigned long long>;
//#else
//using Model = kvgpu::SimplModel<unsigned long long>;
//#endif
using RB = std::shared_ptr<Communication>;

int totalBatches = 10000;
int BATCHSIZE = 512;
int NUM_THREADS = 12;//std::thread::hardware_concurrency() - 10;
std::atomic_bool done(false);
int LOG_LEVEL = WARNING;

void signal_handler(int signo) {
    done = true;
}

void usage(char *command);

struct ServerConf {
    int threads;
    int cpu_threads;

    int gpus;
    int streams;
    std::string modelFile;
    bool train;
    int size;
    int batchSize;
    bool cache;
    std::string address;
    int port;

    ServerConf() {
        batchSize = BATCHSIZE;
        modelFile = "";
        cpu_threads = NUM_THREADS;
        threads = 4;//1;//4;
        gpus = 1;
        streams = 10;//10;
        size = 1000000;
        train = false;
        cache = true;
        address = "127.0.0.1";
        port = 8080;
    }

    explicit ServerConf(const std::string &filename) {
        pt::ptree root;
        pt::read_json(filename, root);
        cpu_threads = root.get<int>("cpu_threads", NUM_THREADS);
        threads = root.get<int>("threads", 4);
        streams = root.get<int>("streams", 2);
        gpus = root.get<int>("gpus", 2);
        modelFile = root.get<std::string>("modelFile", "");
        train = root.get<bool>("train", false);
        size = root.get<int>("size", 1000000);
        batchSize = root.get<int>("batchSize", BATCHSIZE);
        cache = root.get<bool>("cache", true);
        address = root.get<std::string>("address", "127.0.0.1");
        port = root.get<int>("port", 8080);
    }

    void persist(const std::string &filename) const {
        pt::ptree root;
        root.put("threads", threads);
        root.put("streams", streams);
        root.put("gpus", gpus);
        root.put("modelFile", modelFile);
        root.put("train", train);
        root.put("size", size);
        root.put("batchSize", batchSize);
        root.put("cache", cache);
        root.put("address", address);
        root.put("port", port);
        pt::write_json(filename, root);
    }

    ~ServerConf() = default;

};

int main(int argc, char **argv) {

    ServerConf sconf;

    char c;
    while ((c = getopt(argc, argv, "f:")) != -1) {
        switch (c) {
            case 'f':
                sconf = ServerConf(std::string(optarg));
                // optarg is the file
                break;
            default:
            case '?':
                usage(argv[0]);
                return 1;
        }
    }

    std::vector<PartitionedSlabUnifiedConfig> conf;
    for (int i = 0; i < sconf.gpus; i++) {
        for (int j = 0; j < sconf.streams; j++) {
            gpuErrchk(hipSetDevice(i));
            hipStream_t stream = hipStreamDefault;
            if (j != 0) {
                gpuErrchk(hipStreamCreate(&stream));
            }
            conf.push_back({sconf.size, i, stream});
        }
    }

    std::unique_ptr<KVStoreCtx<Model>> ctx = nullptr;
    if (sconf.modelFile != "") {
        ctx = std::make_unique<KVStoreCtx<Model>>(conf, sconf.cpu_threads, sconf.modelFile);
    } else {
        ctx = std::make_unique<KVStoreCtx<Model>>(conf, sconf.cpu_threads);
    }

    GeneralClient<Model> *client = nullptr;
    if (sconf.cache) {
        if (sconf.gpus == 0) {
            client = new JustCacheKVStoreClient<Model>(*ctx);
        } else {
            client = new KVStoreClient<Model>(*ctx);
        }
    } else {
        client = new NoCacheKVStoreClient<Model>(*ctx);
    }

    auto server = new cse498::Connection(sconf.address.c_str(), true, sconf.port);
    bool rerun = false;

    cse498::threadpool clientHandler(sconf.threads);

    if (signal(SIGINT, signal_handler) == SIG_ERR) {
        DO_LOG(ERROR) << "Unable to set signal handler";
        return 1;
    }

    std::thread t = std::thread([&]() {
        while (!done) {

            auto *clientConnection = new cse498::Connection();

            do {
                auto p = server->nonblockingAccept();
                rerun = !p.first;
                if (p.first) {
                    *clientConnection = std::move(p.second);
                }
                if (done)
                    return;
            } while (rerun);
            DO_LOG(TRACE) << "Connection made";

            clientHandler.submit([clientConnection, client]() {
                loadBalanceSet = true;

                uint64_t key = 1;
                auto *buf = new cse498::unique_buf();
                clientConnection->register_mr(*buf, FI_READ | FI_WRITE | FI_SEND | FI_RECV, key);

                std::vector<RequestWrapper<unsigned long long int, data_t *>> clientBatch;

                while (true) {

                    clientConnection->recv(*buf, sizeof(size_t));
                    size_t batchsize = *(size_t *) buf->get();
                    if (batchsize == 0) {
                        delete clientConnection;
                        delete buf;
                        break;
                    }

                    clientBatch.reserve(batchsize);

                    while (clientBatch.size() != batchsize) {
                        clientConnection->recv(*buf, sizeof(size_t));
                        size_t incomingBytes = *(size_t *) buf->get();
                        clientConnection->recv(*buf, incomingBytes);

                        size_t offset = 0;
                        while (offset < incomingBytes) {
                            size_t amountConsumed = 0;
                            auto r = deserialize2<RequestWrapper<unsigned long long, data_t *>>(buf->get() + offset,
                                                                                                incomingBytes,
                                                                                                amountConsumed);
                            clientBatch.push_back(r);
                            offset += amountConsumed;
                        }
                    }

                    auto start = std::chrono::high_resolution_clock::now();
                    std::shared_ptr<Communication> comm = std::make_shared<RemoteCommunication>(clientConnection, buf);
                    DO_LOG(TRACE) << "Batching";
                    client->batch(clientBatch, comm, start);

                    std::cerr << "Ran batch\n";
                }
            });
        }
    });

    // wait on input to end
    while (!done) {
        std::this_thread::yield();
    }

    t.join();
    clientHandler.join();

    delete client;
    //delete server;
    return 0;
}

void usage(char *command) {
    using namespace std;
    cout << command << " [-f <config file>]" << std::endl;
}
