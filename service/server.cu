#include <unistd.h>
#include "helper.cuh"
#include <algorithm>
#include <boost/property_tree/ptree.hpp>
#include <boost/property_tree/json_parser.hpp>
#include <RemoteCommunication.hh>

namespace pt = boost::property_tree;
using BatchWrapper = std::vector<RequestWrapper<unsigned long long, data_t *>>;
//#ifdef MODEL_CHANGE
using Model = kvgpu::AnalyticalModel<unsigned long long>;
//#else
//using Model = kvgpu::SimplModel<unsigned long long>;
//#endif
using RB = std::shared_ptr<Communication>;

int totalBatches = 10000;
int BATCHSIZE = 512;
int NUM_THREADS = 12;//std::thread::hardware_concurrency() - 10;

int LOG_LEVEL = WARNING;

void usage(char *command);

struct ServerConf {
    int threads;
    int cpu_threads;

    int gpus;
    int streams;
    std::string modelFile;
    bool train;
    int size;
    int batchSize;
    bool cache;

    ServerConf() {
        batchSize = BATCHSIZE;
        modelFile = "";
        cpu_threads = NUM_THREADS;
        threads = 2;//1;//4;
        gpus = 1;
        streams = 10;//10;
        size = 1000000;
        train = false;
        cache = true;
    }

    explicit ServerConf(const std::string &filename) {
        pt::ptree root;
        pt::read_json(filename, root);
        cpu_threads = root.get<int>("cpu_threads", NUM_THREADS);
        threads = root.get<int>("threads", 4);
        streams = root.get<int>("streams", 2);
        gpus = root.get<int>("gpus", 2);
        modelFile = root.get<std::string>("modelFile", "");
        train = root.get<bool>("train", false);
        size = root.get<int>("size", 1000000);
        batchSize = root.get<int>("batchSize", BATCHSIZE);
        cache = root.get<bool>("cache", true);
    }

    void persist(const std::string &filename) const {
        pt::ptree root;
        root.put("threads", threads);
        root.put("streams", streams);
        root.put("gpus", gpus);
        root.put("modelFile", modelFile);
        root.put("train", train);
        root.put("size", size);
        root.put("batchSize", batchSize);
        root.put("cache", cache);
        pt::write_json(filename, root);
    }

    ~ServerConf() = default;

};

int main(int argc, char **argv) {

    ServerConf sconf;

    char c;
    while ((c = getopt(argc, argv, "f:")) != -1) {
        switch (c) {
            case 'f':
                sconf = ServerConf(std::string(optarg));
                // optarg is the file
                break;
            default:
            case '?':
                usage(argv[0]);
                return 1;
        }
    }

    std::vector<PartitionedSlabUnifiedConfig> conf;
    for (int i = 0; i < sconf.gpus; i++) {
        for (int j = 0; j < sconf.streams; j++) {
            gpuErrchk(hipSetDevice(i));
            hipStream_t stream = hipStreamDefault;
            if (j != 0) {
                gpuErrchk(hipStreamCreate(&stream));
            }
            conf.push_back({sconf.size, i, stream});
        }
    }

    std::unique_ptr<KVStoreCtx<Model>> ctx = nullptr;
    if (sconf.modelFile != "") {
        ctx = std::make_unique<KVStoreCtx<Model>>(conf, sconf.cpu_threads, sconf.modelFile);
    } else {
        ctx = std::make_unique<KVStoreCtx<Model>>(conf, sconf.cpu_threads);
    }

    GeneralClient<Model> *client = nullptr;
    if (sconf.cache) {
        if (sconf.gpus == 0) {
            client = new JustCacheKVStoreClient<Model>(*ctx);
        } else {
            client = new KVStoreClient<Model>(*ctx);
        }
    } else {
        client = new NoCacheKVStoreClient<Model>(*ctx);
    }

    auto server = new cse498::Connection("127.0.0.1", true, 8080);
    loadBalanceSet = true;
    server->connect();


    uint64_t key = 1;
    auto* buf = new cse498::unique_buf();
    server->register_mr(*buf, FI_READ | FI_WRITE | FI_SEND | FI_RECV, key);

    std::vector<RequestWrapper<unsigned long long int, data_t *>> clientBatch;

    while (true) {

        server->recv(*buf, sizeof(size_t));
        size_t batchsize = *(size_t *) buf->get();
        if (batchsize == 0) {
            break;
        }

        clientBatch.reserve(batchsize);

        while (clientBatch.size() != batchsize) {
            server->recv(*buf, sizeof(size_t));
            size_t incomingBytes = *(size_t *) buf->get();
            server->recv(*buf, incomingBytes);

            size_t offset = 0;
            while (offset < incomingBytes) {
                size_t amountConsumed = 0;
                auto r = deserialize2<RequestWrapper<unsigned long long, data_t *>>(
                        std::vector<char>(buf->get() + offset, buf->get() + incomingBytes), amountConsumed);
                clientBatch.push_back(r);
                offset += amountConsumed;
            }
        }

        auto start = std::chrono::high_resolution_clock::now();
        std::shared_ptr<Communication> comm = std::make_shared<RemoteCommunication>(server, buf);
        client->batch(clientBatch, comm, start);

        std::cerr << "Ran batch\n";
    }

    delete client;
    delete buf;
    //delete server;
    return 0;
}

void usage(char *command) {
    using namespace std;
    cout << command << " [-f <config file>]" << std::endl;
}
