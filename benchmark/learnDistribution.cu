#include "hip/hip_runtime.h"
//
// Created by depaulsmiller on 1/18/21.
//

#include <zipf.hh>
#include <vector>
#include <unordered_map>
#include <map>
#include <cmath>
#include <iostream>

constexpr float sigmoid(float x) {
    return 1.0f / (1.0f + expf(-x));
}

constexpr float d_sigmoid(float x) {
    return sigmoid(x)*(1 - sigmoid(x));
}

constexpr float serr(float y, float ypred) {
    float diff = (y - ypred);
    return diff * diff;
}

constexpr float d_serr(float y, float ypred) {
    float diff = (y - ypred);
    return 2.0f * diff;
}

int main() {

    double theta = 0.99;
    unsigned range = 10000000;
    auto zetaN = betterstd::zeta(theta, range);

    float w = rand() / (float)RAND_MAX;
    float b = 0.0;
    float alpha = 0.1f;

    for(int repeat = 0; repeat < 1000; repeat++) {
        std::vector<int> v;
        std::unordered_map<int, unsigned> m;

        for (int i = 0; i < 512; i++) {
            int gen = betterstd::rand_zipf(range, zetaN, theta);
            if (m.find(gen) != m.end()) {
                m[gen] += 1;
            } else {
                m[gen] = 1;
            }
        }

        std::unordered_map<int, float> pred;
        std::vector<int> input;

        for (auto &p : m) {
            pred[p.first] = p.second / 512.0f;
            input.push_back(p.first);
        }


        std::vector<float> output;
        std::vector<std::pair<float, float>> d_output;

        float sse = 0.0f;

        float d_w = 0.0f;
        float d_b = 0.0f;
        float sumdiff = 0.0f;

        for (auto &x : input) {
            output.push_back(sigmoid(x * w + b));

            auto d_s = d_sigmoid(x * w + b);

            auto p_d_w = x * d_s;
            auto p_d_b = d_s;

            output.push_back(sigmoid(x * w + b));
            d_output.push_back({p_d_w, p_d_b});
        }

        for (int i = 0; i < output.size(); ++i) {
            sumdiff += pred[i] - output[i];
            sse += serr(output[i], pred[i]);
            d_w += d_serr(output[i], pred[i]) * d_output[i].first;
            d_b += d_serr(output[i], pred[i]) * d_output[i].second;
        }

        std::cerr << alpha * d_w << " " << sumdiff << std::endl;
        std::cerr << alpha * d_b << " " << sumdiff << std::endl;

        w -= alpha * d_w;
        b -= alpha * d_b;

        std::cout << sse << std::endl;
    }
    return 0;
}